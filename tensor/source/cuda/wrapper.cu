//
// Copyright (C) Chris Zankel. All rights reserved.
// This code is subject to U.S. and other copyright laws and
// intellectual property protections.
//
// The contents of this file are confidential and proprietary to Chris Zankel.
//

#include <stdexcept>

#include <grid/tensor/cuda/array.h>

namespace grid {

void CudaMalloc(void** ptr, size_t size)
{
  auto err = hipMalloc(ptr, size);
  if (err != hipSuccess)
    throw std::runtime_error(std::string("hipMalloc failed") + hipGetErrorString(err));
}

void CudaMallocManaged(void** ptr, size_t size)
{
  auto err = hipMallocManaged(ptr, size);
  if (err != hipSuccess)
    throw std::runtime_error(std::string("hipMallocManaged failed") + hipGetErrorString(err));
}


void CudaFree(void* ptr)
{
  auto err = hipFree(ptr);
  if (err != hipSuccess)
    throw std::runtime_error(std::string("hipMalloc failed") + hipGetErrorString(err));
}

void CudaDeviceSynchronize()
{
  auto err = hipDeviceSynchronize();
  if (err != hipSuccess)
    throw std::runtime_error(std::string("hipMalloc failed") + hipGetErrorString(err));
}

} // end of namespace grid
