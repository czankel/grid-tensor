#include "hip/hip_runtime.h"
//
// Copyright (C) Chris Zankel. All rights reserved.
// This code is subject to U.S. and other copyright laws and
// intellectual property protections.
//
// The contents of this file are confidential and proprietary to Chris Zankel.
//

#include <grid/tensor/tensor.h>
#include <grid/tensor/precision.h>

#include <grid/tensor/cuda/device.h>
#include <grid/tensor/cuda/softmax.h>

#include "../instantiate.h"
#include "reduce.h"
#include "utils.h"

namespace grid {
namespace cuda {

template <typename T, unsigned int BlockSize>
__global__ void CudaSoftMax(T* d, const T* x, T eps, int dim)
{
  __shared__ T sdata[cuda::MaxThreadCount];

  unsigned int grid_size = gridDim.x * BlockSize;
  unsigned int tid = threadIdx.x;

  T max{0};  // FIXME: use T or float?
  // FIXME unroll (partially)
  for (unsigned int i = tid; i < dim; i += grid_size)
    max = x[i] > max ? x[i] : max;
  sdata[tid] = max;

  __syncthreads();

  CudaReduce<T, MaxOperator, BlockSize>(sdata, tid, dim);
  max = sdata[0];

  // d = exp(x - local-max); sum += d
  T sum{0};
  for (unsigned int i = tid; i < dim; i += grid_size)
  {
    d[i] = exp(x[i] - max);
    sum += d[i];
  }
  sdata[tid] = sum;

  __syncthreads();

  CudaReduce<T, AddOperator, BlockSize>(sdata, tid, dim);
  sum = sdata[0];
  T scale = static_cast<T>(1)/(sum + eps);
  printf("max %f sum %f\n", max, sum);

  __syncthreads();

  for (unsigned int i = tid; i < dim; i += grid_size)
    d[i] = d[i] * scale;
}

#if 0
template <typename T, unsigned int BlockSize>
__global__ void CudaSoftMax(T* d, const T* x, const T eps, int dim)
{
  __shared__ T sdata[cuda::MaxThreadCount];

  unsigned int tid_col = threadIdx.x;
  unsigned int tid_row = threadIdx.y * dim;
  unsigned int tid = tid_col + tid_row;
  unsigned int end = tid_col + dim;

  unsigned size_t block_start = blockIdx.y * blockDim.x + blockIdx.x * BlockSize;
  unsigned int grid_size = gridDim.x * BlockSize;
  x = x + block_start;

  T max{0};  // FIXME: use T or float?
  // FIXME unroll (partially)
  for (unsigned int i = tid; i < end; i += grid_size)
    max = std::max(max, x[i]);
  sdata[tid] = max;

  __syncthreads();

  max = CudaMapReduce(sdata, tid, end);

  // FIXME __syncthreads();

  // SumExp
  // exp(x - local-max)
  for (unsigned int i = idx_s; i < idx_e; i += grid_size)
  {
    d[i] = exp(x[i] - max);
    sdata[tid] += d[i];
  }

  //T sum = CudaAllReduce(sdata, tid_x, dim);
  //T scalar = static_cast<T>(1)/(sum + eps);

  //for (unsigned int i = idx_s; i < idx_e; i += grid_size)
    //d[i] = x[i] / scalar;
}
#endif

} // end of namespace cuda

template <typename T>
void SoftMaxCallKernel(T* d, const T* x, size_t rows, size_t cols, dim3 grid_size, dim3 block_size)
{
  size_t smem_size = cuda::MaxThreadCount;
  size_t n_threads = std::min(cols, 1024UL);  // FIXME: divide by sizeof(T) *2??
  int n_threads_log2 = sizeof(n_threads) * 8 - __builtin_clzll(n_threads - 1);

  T eps = Eps<T>::default_value;
printf("n-thread %d\n", n_threads_log2);
  switch (n_threads_log2)
  {
    #define CUDA_SOFTMAX_CASE(BIT) \
      case BIT: cuda::CudaSoftMax<T,1<<BIT><<<grid_size,block_size,smem_size>>>(d,x,eps,cols); break;
    INSTANTIATE1(CUDA_SOFTMAX_CASE, (9, 8, 7, 6, 5, 4, 3, 2, 1, 0))
    default: throw std::runtime_error("invalid thread count in softmax");
  }
}


// note that lower ranks are contiguous
template <typename T, size_t R>
void SoftMaxOperator<device::Cuda>::EvalContiguous(
    T* d, const T* x,
    std::span<const size_t, R> dimensions,
    std::span<const ssize_t, R> strides_d,
    std::span<const ssize_t, R> strides_a) const
{
  if constexpr (R == 0)
  {
    printf("SF1\n");
    SoftMaxCallKernel(d, x, 1, 1, 1, 1);
  }
  else if constexpr (R == 1)
  {
    printf("SF2\n");
    auto [grid_size, block_size] = cuda::GetSizes(dimensions[0]);
    SoftMaxCallKernel(d, x, 1, dimensions[0], std::move(grid_size), std::move(block_size));
  }
  else if constexpr (R == 2)
  {
    printf("OPS1\n");
#if 0
    auto [block_size, grid_size] = cuda::GetSizes(dimensions, 16, 16);  // FIXME 256 threads instead of 1k? 32, 32?
    CudaSoftMaxContiguousRank2<T><<<block_size, grid_size>>>(
        d, a, cuda::MakeDim3(dimensions), cuda::MakeDim3(strides_d), cuda::MakeDim3(strides_a));
#endif
  }
  else if constexpr (R == 3)
  {
    printf("OPS1\n");
#if 0
    auto [block_size, grid_size] = cuda::GetSizes(dimensions, 8, 8, 8);   // FIXME 512 threads?
    CudaSoftMaxContiguousRank3<T><<<block_size, grid_size>>>(
        d, a, cuda::MakeDim3(dimensions), cuda::MakeDim3(strides_d), cuda::MakeDim3(strides_a));
#endif
  }
}


template <typename T, size_t R>
void SoftMaxOperator<device::Cuda>::EvalDiscontiguous(
    T* d, const T* a,
    std::span<const size_t, R> dimensions,
    std::span<const ssize_t, R> strides_d,
    std::span<const ssize_t, R> strides_a) const
{
  printf("DISCONTIGUOUS@!!\n");
}

#define FUNCTION_CONTIGUOUS(R, T) \
  template void SoftMaxOperator<device::Cuda>::EvalContiguous<T, R>( \
      T*, const T*, std::span<const size_t, R>, \
      std::span<const ssize_t, R>, std::span<const ssize_t, R>) const;

#define FUNCTION_DISCONTIGUOUS(R, T) \
  template void SoftMaxOperator<device::Cuda>::EvalDiscontiguous<T, R>( \
      T*, const T*,  std::span<const size_t, R>, \
      std::span<const ssize_t, R>, std::span<const ssize_t, R>) const;

#define TYPES  float
#define RANKS_CONTIGUOUS 0, 1, 2, 3
#define RANKS_DISCONTIGUOUS 1, 2, 3

INSTANTIATE2(FUNCTION_CONTIGUOUS, (RANKS_CONTIGUOUS), (TYPES))
INSTANTIATE2(FUNCTION_DISCONTIGUOUS, (RANKS_DISCONTIGUOUS), (TYPES))

} // end of namespace grid
