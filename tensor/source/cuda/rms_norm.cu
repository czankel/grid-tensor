#include "hip/hip_runtime.h"
//
// Copyright (C) Chris Zankel. All rights reserved.
// This code is subject to U.S. and other copyright laws and
// intellectual property protections.
//
// The contents of this file are confidential and proprietary to Chris Zankel.
//

#include <utility>

#include <grid/tensor/tensor.h>

#include <grid/tensor/cuda/binary.h>
#include <grid/tensor/cuda/device.h>

#include <grid/tensor/cuda/rms_norm.h>

#include "../instantiate.h"
#include "utils.h"


namespace grid {
namespace cuda {

template <typename T, unsigned int BlockSize>
__device__ void CudaWarpReduce(volatile T* sdata, unsigned int tid, unsigned int dim)
{ // FIXME: tid vs tid_x for < dim??
  if (BlockSize >= 64 && tid + 32 < dim) sdata[tid] = sdata[tid] + sdata[tid + 32];
  if (BlockSize >= 32 && tid + 16 < dim) sdata[tid] = sdata[tid] + sdata[tid + 16];
  if (BlockSize >= 16 && tid +  8 < dim) sdata[tid] = sdata[tid] + sdata[tid +  8];
  if (BlockSize >=  8 && tid +  4 < dim) sdata[tid] = sdata[tid] + sdata[tid +  4];
  if (BlockSize >=  4 && tid +  2 < dim) sdata[tid] = sdata[tid] + sdata[tid +  2];
  if (BlockSize >=  2 && tid +  1 < dim) sdata[tid] = sdata[tid] + sdata[tid +  1];
}

template <typename T, unsigned int BlockSize>
__global__ void CudaRmsNorm(T* d, const T* x, const T eps, int dim)
{
  __shared__ T sdata[cuda::MaxThreadCount];

  unsigned int tid_x = threadIdx.x;
  unsigned int lane = threadIdx.y * dim;
  unsigned int tid = lane + tid_x;
  unsigned int idx_s = blockIdx.x * BlockSize + tid;
  unsigned int idx_e = lane + dim;
  unsigned int grid_size = gridDim.x * BlockSize;

  T sum{0};
  for (unsigned int i = idx_s; i < idx_e; i += grid_size)
    sum += x[i] * x[i];

  sdata[tid] = sum;

  __syncthreads();

  if (BlockSize >= 512) { if (tid_x < 256) { sdata[tid] += sdata[tid + 256]; } __syncthreads(); }
  if (BlockSize >= 256) { if (tid_x < 128) { sdata[tid] += sdata[tid + 128]; } __syncthreads(); }
  if (BlockSize >= 128) { if (tid_x < 64)  { sdata[tid] += sdata[tid + 64];  } __syncthreads(); }
  if (tid_x < 32) CudaWarpReduce<T, BlockSize>(sdata, tid, lane + dim);

  if (tid_x == 0)
    sdata[lane] = sqrt(sdata[lane] / dim + eps);

  __syncthreads();

  for (unsigned int i = idx_s; i < idx_e; i += grid_size)
    d[i] = x[i] / sdata[lane];
}

} // end of namespace cuda


template <typename T>
void RmsNormOperator<device::Cuda>::Eval(T* d, const T* x, const T eps, size_t rows, size_t cols) const
{
  const size_t dims[]{cols, rows};
  size_t warp_size = static_cast<size_t>(cuda::WarpSize); // FIXME:

  // use as many parallel threads (rows) as we can
  size_t blk_x = ((rows + warp_size - 1) / warp_size) * warp_size; // FIXME: why align up to warp_size??
  size_t blk_y = std::min(cuda::MaxThreadCount / blk_x, rows);
  blk_x = std::min(blk_x, cols);

  auto [grid_size, block_size] = cuda::GetSizes(std::span{dims}, blk_x, blk_y);

  size_t smem_size = cuda::MaxThreadCount;
  size_t n_threads = cols > 1024 ? 1024 : cols; // FIXME
  int n_threads_log2 = sizeof(n_threads) * 8 - __builtin_clzll(n_threads - 1);

  switch (n_threads_log2)
  {
    #define CUDA_RMS_NORM_CASE(BIT) \
      case BIT: cuda::CudaRmsNorm<T,1<<BIT><<<grid_size,block_size,smem_size>>>(d,x,eps,cols); break;
    INSTANTIATE1(CUDA_RMS_NORM_CASE, (9, 8, 7, 6, 5, 4, 3, 2, 1, 0))
    default: throw std::runtime_error("invalid thread count");
  }
}

#define FUNCTION(T) \
  template void RmsNormOperator<device::Cuda>::Eval<T>(T*, const T*, const T, size_t, size_t) const; 

#define TYPES  float

INSTANTIATE1(FUNCTION, (TYPES))

} // end of namespace grid
